#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>
#include <stdlib.h>


#include "macros.h"



unsigned long Time = 0;



__global__ void ReduceNeighbored( int *g_array, int *g_output, int arrayLength ){

   int tid = blockIdx.x*blockDim.x + threadIdx.x;

   if (tid >= arrayLength) return;

   int *idata = g_array + blockIdx.x*blockDim.x;

   for ( int offset = 1; offset < blockDim.x; offset *= 2 ){

      if (threadIdx.x % ( 2*offset ) == 0){
         idata[threadIdx.x] += idata[threadIdx.x+offset];
      }
      __syncthreads();
   }

   if (threadIdx.x == 0) g_output[blockIdx.x] = g_array[blockIdx.x*blockDim.x];

}


__global__ void ReduceNeighboredLess( int *g_array, int *g_output, int arrayLength ){

   int tid = blockIdx.x*blockDim.x + threadIdx.x;

   if (tid >= arrayLength) return;

   int *idata = g_array + blockIdx.x*blockDim.x;

   for ( int offset = 1; offset < blockDim.x; offset *= 2 ){

      if ( threadIdx.x < blockDim.x/2/offset ){
         idata[threadIdx.x*offset*2] += idata[threadIdx.x*offset*2 + offset];
      }
      __syncthreads();
   }

   if (threadIdx.x == 0) g_output[blockIdx.x] = g_array[blockIdx.x*blockDim.x];

}


// warp divergence only occurs in the last five rounds
// when the number of active threads is less than a warp size (32)

__global__ void ReduceInterleaved( int *g_array, int *g_output, int arrayLength ){

   int tid = blockIdx.x*blockDim.x + threadIdx.x;

   if (tid >= arrayLength) return;

   int *idata = g_array + blockIdx.x*blockDim.x;

   for ( int offset = blockDim.x/2; offset > 0; offset /= 2 ){

      if (threadIdx.x < offset){
         idata[threadIdx.x] += idata[threadIdx.x+offset];
      }
      __syncthreads();
   }

   if (threadIdx.x == 0) g_output[blockIdx.x] = g_array[blockIdx.x*blockDim.x];

   // The line below shows "1024 4 4 4 4 4 4 4" since the blocks with block IDs greater than 1
   // will access g_array[] before the 0th block
   //if (threadIdx.x == 0) g_output[blockIdx.x] = g_array[threadIdx.x];

}


__global__ void ReduceUnrolling2( int *g_array, int *g_output, int arrayLength ){

   int *idata = g_array + 2*blockIdx.x*blockDim.x;

   if ( threadIdx.x + blockDim.x < arrayLength ){
      int a0 = idata[threadIdx.x           ];
      int a1 = idata[threadIdx.x+blockDim.x];
      idata[threadIdx.x] = a0 + a1;
   }

   __syncthreads();

   for ( int offset = blockDim.x/2; offset > 0; offset /= 2 ){

      if (threadIdx.x < offset){
         idata[threadIdx.x] += idata[threadIdx.x+offset];
      }
      __syncthreads();
   }

   if (threadIdx.x == 0) g_output[blockIdx.x] = idata[0];
}


__global__ void ReduceUnrolling3( int *g_array, int *g_output, int arrayLength ){

   int *idata = g_array + 3*blockIdx.x*blockDim.x;

   if ( threadIdx.x + 2*blockDim.x < arrayLength ){
      int a0 = idata[threadIdx.x             ];
      int a1 = idata[threadIdx.x+  blockDim.x];
      int a2 = idata[threadIdx.x+2*blockDim.x];
      idata[threadIdx.x] = a0 + a1 + a2;
   }

   __syncthreads();


   for ( int offset = blockDim.x/2; offset > 0; offset /= 2 ){

      if (threadIdx.x < offset){
         idata[threadIdx.x] += idata[threadIdx.x+offset];
      }

      __syncthreads();
   }

   if (threadIdx.x == 0) g_output[blockIdx.x] = idata[0];
}

int sum( int *array, int arrayLength ){

   int sum = 0;

   for ( int i=0; i<arrayLength; i++ ){
      sum += array[i];
   }

   return sum;
}



int main(){

   int arrayLength = 128*28;

   int blockSize = 128;

   dim3 block ( blockSize, 1 );
   dim3 grid ( ( arrayLength + block.x - 1 ) / block.x, 1 );


   int *h_array, *g_array, *g_output, *h_output = NULL;

   h_array  = (int*)calloc(arrayLength, sizeof(int));
   h_output = (int*)calloc(     grid.x, sizeof(int));

   int max = +100;
   int min =  +10;
   srand(time(NULL));

   for (int i=0; i<arrayLength; i++) h_array[i] = min + (rand() % (max - min + 1));

   CHECK(  hipMalloc( &g_array, arrayLength*sizeof(int) )  );
   CHECK(  hipMalloc( &g_output,     grid.x*sizeof(int) )  );

   CHECK( hipMemcpy( g_array      , h_array      , arrayLength*sizeof(int), hipMemcpyHostToDevice ) );


   BlockReduction4<<< grid.x/2, block, 0, 0 >>>
   ( g_array, g_output, arrayLength );

   CHECK( hipMemcpy( h_output, g_output, grid.x*sizeof(int), hipMemcpyDeviceToHost ) );


   // sum up partial sum from thread blocks
   int g_sum = 0;

   for (int i=0; i<grid.x; i++) g_sum += h_output[i];


   if ( g_sum == sum(h_array, arrayLength) ) printf("Pass!\n");
   else                                      printf("Fail!\n");



   return 0;
}
